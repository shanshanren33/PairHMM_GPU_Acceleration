	#include <iostream>
	#include <stdlib.h>
	#include <stdio.h>
	#include <string.h>
	#include <time.h>
	#include <hip/hip_runtime.h>
	#include <stdint.h>
	#include <math.h>
	#include <unistd.h>
	#include <omp.h>	
	#include <algorithm>
	using namespace std;

	// 8 byte.   how to be 128byte?
	// Parameter need to restruct.
	//2 bytes, 2 bytes, 4 bytes, 4 bytes, 4 bytes.
	struct NUM_ADD
	{
		short2 read_haplotype;
		int  Read_array;
		int read_large_length;
	};

	double diff(timespec start, timespec end)
	{
	  double a=0;
	 if((end.tv_nsec-start.tv_nsec)<0)
	{
	a=end.tv_sec-start.tv_sec-1;
	a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
	}
	else
	{
	a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

	}
	return a;
	}

	__constant__ float  constant[10];
	__constant__ int  constant_int[10];
	

	__global__ void  pairHMM( int size, char * data,  NUM_ADD * num_add, float * result,float * MG,float * DG, float * IG ) // what is the maximum number of parameters?
	{
	//MG, DG and IG are global memory to store indermediate result?
	//each thread finish one computation		
	int offset=blockIdx.x*blockDim.x+threadIdx.x;
	MG=MG+offset;
	IG=IG+offset;
	DG=DG+offset;
	//if(threadIdx.x==0)
	//printf("%d %d %d %d %d\n", constant_int[0],constant_int[1], constant_int[2],constant_int[3], constant_int[4]);
	while(offset<size)
	 {	
	
		//NUM_ADD number_address;
		//number_address=num_add[offset];//get from global memory
		short2 read_haplotype_number=num_add[offset].read_haplotype;
		int read_large_length=num_add[offset].read_large_length;
		//read_haplotype_number.x=number_address.read_number;	
		char4 * read_base_array=(char4 *)(data+num_add[offset].Read_array); // to caculate the address of read_base_array. 
		float  *parameter1_array=(float *) (read_base_array+(read_large_length+3)/4*32);
		read_large_length=read_large_length*32;
		float  *parameter2_array=(float *) (parameter1_array+read_large_length);
		float  *parameter3_array=(float *) (parameter1_array+read_large_length*2);
		float  *parameter4_array=(float *) (parameter1_array+read_large_length*3);
		//read_haplotype_number.y=number_address.haplotype_number;
		char4 * haplotype_base_array=(char4 * )(parameter1_array+read_large_length*4);    	
		//haplotype is 4 byte. Thus, in a warp it is 4*32=128 byte. //we need to change the struct of haplotype

		float  result_block;
		result_block=constant[5];
	

		int i;
	//	__shared__ float delta[128];
	//	__shared__ float xiksi[128];
	//	__shared__ float alpha[128];
	//	__shared__ float Qm[128];
		//try to use share_memory to store paramers. 
                 char4 read_base_4;
		for(i=0;i<read_haplotype_number.x;i++)
                {
                if(i%4==0)
                {
                        read_base_4=read_base_array[i/4*constant_int[2]];
                }
                char2 read_haplotype_base;
                if(i%4==0) read_haplotype_base.x=read_base_4.x;
                if(i%4==1) read_haplotype_base.x=read_base_4.y;
                if(i%4==2) read_haplotype_base.x=read_base_4.z;
                if(i%4==3) read_haplotype_base.x=read_base_4.w;

         	float Qm,Qm_1,alpha,delta,xiksi;
			
		Qm=parameter1_array[i*constant_int[2]];	
		delta=parameter2_array[i*constant_int[2]];
		Qm_1=constant[1]-Qm;
		xiksi=parameter3_array[i*constant_int[2]];
		alpha=parameter4_array[i*constant_int[2]];
		Qm=fdividef(Qm,constant[2]);
		//load all the data into shared memory or registers.
                float Ml=0;// left M;
                float Dl=0;// left D;
                float Il=0;
                float MU=0;// up M;
                float IU=0;// up I;
                float DU=0;// up D;
                float MMID=0;

		 if(i==0)
                {
                DU=constant[0]/(float) read_haplotype_number.y;
                MMID=__fmul_rn(constant[3],DU);
                }

                int hh=(read_haplotype_number.y+4-1)/4;
                for(int j=0;j<hh;j++)
                {
                char4 haplotype_base;
                haplotype_base=haplotype_base_array[j*constant_int[2]];

                for(int kk=0;kk<4;kk++)
                {
                                if(j*4+kk==read_haplotype_number.y)
                                        break;
				if(kk==0)
                                       read_haplotype_base.y=haplotype_base.x;
                                if(kk==1)
                                        read_haplotype_base.y=haplotype_base.y;

				 if(kk==2)
                                        read_haplotype_base.y=haplotype_base.z;
                                if(kk==3)
                                        read_haplotype_base.y=haplotype_base.w;


                                int index=(j*4+kk)*blockDim.x*gridDim.x;
                                if(i>0)
                                {
                                        //here should not using offset. But using the
                                        //get MU,IU,DU from global memory
                                        MU=MG[index];
                                        IU=IG[index];
                                        DU=DG[index];
                                }
 			
                                float MID=__fadd_rn(IU,DU);
                                float DDM=__fmul_rn(Ml,xiksi);
                                float IIMI=__fmul_rn(IU,constant[4]);
                                float aa=(read_haplotype_base.y==read_haplotype_base.x)? Qm_1:Qm;

                                float MIIDD=__fmul_rn(constant[3],MID);
                                Ml=__fmul_rn(aa,MMID);
                                Il=__fmaf_rn(MU,delta,IIMI);
                                Dl=__fmaf_rn(Dl,constant[4],DDM);

                                MMID=__fmaf_rn(alpha,MU,MIIDD);

                                if(i<read_haplotype_number.x-1)
                                {
                                MG[index]=Ml;
                                IG[index]=Il;
                                DG[index]=Dl;
                                }
                                else
                                        result_block=__fadd_rn(result_block,__fadd_rn(Ml,Il));
                        }//4
                } //haplotype

                }//read
		result[offset]=result_block;
		offset+=gridDim.x*blockDim.x ;	
	 }

}


struct InputData
{
int read_size;
char read_base[150];
char base_quals[150];
char ins_quals[150];
char del_quals[150];
char gcp_quals[150];
int haplotype_size;
char haplotype_base[500];
};

bool operator<(const InputData &a, const InputData &b)
{
 //   return x.point_value > y.point_value;
	if(a.read_size<b.read_size) return true;
	if(a.read_size==b.read_size) return a.haplotype_size<b.haplotype_size;
	else
	return false;
	
}




int main(int argc, char * argv[])
{
		//printf("input value of size_each_for \n");
		//scanf("%d", &size_each_for);
		struct timespec start,finish;
		double  computation_time=0,mem_cpy_time=0,read_time=0, data_prepare=0;
		double total_time=0;
		FILE * file;
	//	file=fopen("pairHMM_input_store.txt","r");
		file=fopen(argv[1],"r");
		//file=fopen("32_data.txt","r");
	//	file=fopen("less.txt","r");
		int size;
		fscanf(file,"%d",&size);

		clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
		float ph2pr_h[128];
		for(int i=0;i<128;i++)
		{
			ph2pr_h[i]=powf(10.f, -((float)i) / 10.f);
		}
		hipError_t err;
		
		int  constants_h_int[10];
		float constants_h[10];
		constants_h[0]=1.329228e+36;
		constants_h[1]=1.0;
		constants_h[2]=3.0;
		constants_h[3]=0.9;
		constants_h[4]=0.1;
		constants_h[5]=0.0;
		constants_h_int[0]=0;
		constants_h_int[1]=128;
		constants_h_int[2]=32;
		constants_h_int[3]=4;
		constants_h_int[4]=3;

		hipMemcpyToSymbol(HIP_SYMBOL(constant),constants_h,sizeof(float)*10 );
		hipMemcpyToSymbol(HIP_SYMBOL(constant_int),constants_h_int,sizeof(int)*10 );
			
	
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);	
		data_prepare+=diff(start,finish);
		
		int total=0;
		int fakesize=500000;
		char * result_d_total;
		float read_read, haplotype_haplotype;
		while(!feof(file))
		{
			total+=size;
			char useless;
			useless=fgetc(file);
			
			clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
			
			InputData *inputdata=(InputData* )malloc(fakesize*(sizeof(InputData)));		
			for(int i=0;i<size;i++)
			{
				int read_size;
				fscanf(file,"%d\n",&inputdata[i].read_size);
				fscanf(file,"%s ",inputdata[i].read_base);
				read_size=inputdata[i].read_size;
				read_read=read_size;
			
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i]. base_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].ins_quals[j]=(char)aa;
				}
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].del_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				if(j<read_size-1) fscanf(file,"%d ",&aa);
				else  fscanf(file,"%d \n",&aa);
				 inputdata[i].gcp_quals[j]=(char)aa;
				}

				fscanf(file,"%d\n",&inputdata[i].haplotype_size);
				fscanf(file, "%s\n",inputdata[i].haplotype_base);
				haplotype_haplotype=inputdata[i].haplotype_size;
			}
			clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			read_time+=diff(start,finish);
			
			size=fakesize;
   			float * result_h=(float *) malloc(sizeof(float)*size);
			for(int i=1;i<fakesize;i++)
			{	
				inputdata[i].read_size=inputdata[0].read_size;
				memcpy(inputdata[i].read_base, inputdata[0].read_base,inputdata[0].read_size);
				for(int j=0;j<inputdata[0].read_size;j++)
				{
					inputdata[i].base_quals[j]=inputdata[0].base_quals[j];
					inputdata[i].ins_quals[j]=inputdata[0].ins_quals[j];
					inputdata[i].del_quals[j]=inputdata[0].del_quals[j];
					inputdata[i].gcp_quals[j]=inputdata[0].gcp_quals[j];
				}
				inputdata[i].haplotype_size=inputdata[0].haplotype_size;
				memcpy(inputdata[i].haplotype_base, inputdata[0].haplotype_base,inputdata[0].haplotype_size);
			}
			struct timespec start_total,finish_total;
			clock_gettime(CLOCK_MONOTONIC_RAW,&start_total); 
			char * data_h_total;
							
			 std::sort(inputdata, inputdata+size);
			
			//32 one chunck.
			int malloc_size_for_each_chunk=(40*4*32+150*4*32*4+50*4*32) ;
			int total_size=(size+31)/32*malloc_size_for_each_chunk+(size*sizeof(NUM_ADD)+127)/128*128;
			data_h_total=(char*)malloc(total_size);
			err=hipMalloc( (char **) &result_d_total,total_size+size*sizeof(float));
        		if(err!=hipSuccess)
                        printf("CUDA Error %d:%s !\n", err, hipGetErrorString(err));
			char * data_d_total=result_d_total;
	             	float * result_d=(float *)(result_d_total+total_size);//last part is to store the result.     

			char * data_h=data_h_total;
			char * data_h_begin=data_h; 
			NUM_ADD *data_num_add=(NUM_ADD *) (data_h);
			
			data_h=data_h+(size*sizeof(NUM_ADD)+127)/128*128; // it is 64*x .thus we donot need to worry about alignment.
			int data_size=0;
		
			//for each chunk
			int total_in_each=(size+31)/32;
			for(int i=0;i<total_in_each;i++)
			{
			//each is 32 
			//printf("total_in_each %d\n",total_in_each);
			//read_base
			int long_read_size=0;
			//to find the longest read_size
			for(int j=0;j<32;j++)
			{
			if(i*32+j>=size)
				break;
			if(long_read_size<inputdata[i*32+j].read_size)
				long_read_size=inputdata[i*32+j].read_size;
			}

			int change_length=(long_read_size+3)/4;//because tile=4; each time deal with 4 read
			char4 read_base_data[32*50];
			for(int kk=0;kk<change_length;kk++)
			{
				for(int dd=0;dd<32;dd++) //
				{
					if(i*32+dd>=size)
						break;

					if(inputdata[i*32+dd].read_size<=kk*4)
						continue;
					else
					read_base_data[kk*32+dd].x=inputdata[i*32+dd].read_base[kk*4];
				
					if(inputdata[i*32+dd].read_size<=kk*4+1)
						continue;
					else
					read_base_data[kk*32+dd].y=inputdata[i*32+dd].read_base[kk*4+1];
					
					if(inputdata[i*32+dd].read_size<=kk*4+2)
						continue;
					else
					read_base_data[kk*32+dd].z=inputdata[i*32+dd].read_base[kk*4+2];
				
					if(inputdata[i*32+dd].read_size<=kk*4+3)
						continue;
					else
					read_base_data[kk*32+dd].w=inputdata[i*32+dd].read_base[kk*4+3];
				}
			}	
			//finish read_base

			float parameter1[150*32];//Qm//128 do not change to 128
			float parameter2[150*32];//QI//128 do not change to 128
			float parameter3[150*32];//QD/128 do not change to 128
			float parameter4[150*32];//alpha//128 do not change to 128
			for(int kk=0;kk<long_read_size;kk++)
			{
				for(int dd=0;dd<32;dd++)
				{
					if(i*32+dd>=size)
						break;
					
					if(inputdata[i*32+dd].read_size<=kk)
						continue;
					else
					{
					parameter1[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].base_quals[kk]&127];   
					parameter2[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].ins_quals[kk]&127]  ;
					parameter3[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].del_quals[kk]&127] ;
					parameter4[kk*32+dd]= 1.0f-ph2pr_h[((int)(inputdata[i*32+dd].ins_quals[kk]&127)+(int)( inputdata[i*32+dd].del_quals[kk]&127))&127];
		//			printf("kk=%d  x=%d  y=%d z=%d w=%d \n ",kk,parameter1[kk*32+dd],parameter2[kk*32+dd],parameter3[kk*32+dd],parameter4[kk*32+dd] );
					}		
				}
			}
			
			//to haplotype into 32 char4
			int long_haplotype_size=0;
			//to find the longest hapltoype_size
			for(int j=0;j<32;j++)
			{
			if(i*32+j>=size)
				break;
			if(long_haplotype_size<inputdata[i*32+j].haplotype_size)
				long_haplotype_size=inputdata[i*32+j].haplotype_size;
			}

			int haplotype_change_length=(long_haplotype_size+3)/4;
			char4 haplotype_base_data[125*32];
			for(int kk=0;kk<haplotype_change_length;kk++)
			{
				for(int dd=0;dd<32;dd++)
				{
					if(i*32+dd>=size)
						break;
					if(inputdata[i*32+dd].haplotype_size<=kk*4)
						continue;
					else
					haplotype_base_data[kk*32+dd].x=inputdata[i*32+dd].haplotype_base[kk*4];
				
					if(inputdata[i*32+dd].haplotype_size<=kk*4+1)
						continue;
					else
					haplotype_base_data[kk*32+dd].y=inputdata[i*32+dd].haplotype_base[kk*4+1];
					
					if(inputdata[i*32+dd].haplotype_size<=kk*4+2)
						continue;
					else
					haplotype_base_data[kk*32+dd].z=inputdata[i*32+dd].haplotype_base[kk*4+2];
				
					if(inputdata[i*32+dd].haplotype_size<=kk*4+3)
						continue;
					else
					haplotype_base_data[kk*32+dd].w=inputdata[i*32+dd].haplotype_base[kk*4+3];
				}
			}

			//put data address to each pair of read and haplotype.
			// read address
			memcpy(data_h,read_base_data,sizeof(char4)*32*change_length);//128
			for(int kk=0;kk<32;kk++)
			{
				if(i*32+kk>=size) break;
				data_num_add[i*32+kk].read_haplotype.x=inputdata[i*32+kk].read_size;
				data_num_add[i*32+kk].read_haplotype.y=inputdata[i*32+kk].haplotype_size;
				data_num_add[i*32+kk].Read_array=data_size+sizeof(char4)*kk;
		//		printf("set read size %d %d \n", data_num_add[i*32+kk].read_number,data_num_add[i*32+kk].haplotype_number);
			}

			data_h+=sizeof(char4)*32*change_length;
			data_size+=sizeof(char4)*32*change_length;
			
			//parameter address
			memcpy(data_h,parameter1,sizeof(float)*32*long_read_size);
			for(int kk=0;kk<32;kk++)
			{
				if(i*32+kk>=size) break;
				data_num_add[i*32+kk].read_large_length=long_read_size;
			}
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
			
			memcpy(data_h,parameter2,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		
			memcpy(data_h,parameter3,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		
			memcpy(data_h,parameter4,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		

			//haplotype address
			memcpy(data_h,haplotype_base_data,sizeof(char4)*32*haplotype_change_length);
			data_h+=sizeof(char4)*32*haplotype_change_length;
			data_size+=sizeof(char4)*32*haplotype_change_length;
			}
				
			int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;			
			char * data_d;
			NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
			data_d=data_d_total+(sizeof(NUM_ADD)*size+127)/128*128;
			//printf("data_d_total  %p   num_add_d  %p     data_d %p \n",data_d_total,  num_add_d,data_d);		
			int blocksize=128;
			int gridsize=135; //150;
			dim3 block(blocksize);
			dim3 grid(gridsize);
			// global memory to be used by GPU kernels.
			float * MG;
			float * DG;
			float * IG;
			
			err=hipMemcpy(data_d_total,data_h_begin,data_size_to_copy,hipMemcpyHostToDevice);
			if(err!=hipSuccess)
			printf("cuda Error %d: %s !\n", err, hipGetErrorString(err));
			hipMalloc( (float **)& MG,sizeof(float) *blocksize*gridsize*500*3);
			DG=MG+blocksize*gridsize*500;// ????
			IG=DG+blocksize*gridsize*500;  //?????
		 
			clock_gettime(CLOCK_MONOTONIC_RAW,&start);
			pairHMM<<<grid,block>>> (size,data_d,num_add_d, result_d,MG,DG,IG);
                        hipMemcpy(result_h,result_d,size*sizeof(float),hipMemcpyDeviceToHost);
                       	clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
    		
			computation_time+=diff(start,finish);
		    	
			for(int i=0;i<1;i++)
		   	printf("  i=%d  %e\n",i, result_h[i]);
		
			free(result_h);
			free(data_h_total);
         		hipFree(result_d_total);
	//		
			
                       	clock_gettime(CLOCK_MONOTONIC_RAW,&finish_total);
			total_time+=diff(start_total,finish_total);		
			free(inputdata);
			fscanf(file,"%d",&size);
	//	if(total>10000)
	//		break;
		}
		
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
    
	 	hipDeviceReset();
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
		mem_cpy_time+=diff(start,finish);//(finish1.tv_nsec-start1.tv_nsec)/1000000000.0;

		printf("read_time=%e  initial_time=%e  computation_time= %e total_time=%e\n",read_time, data_prepare,computation_time, total_time);
		printf("GCUPS: %lf \n",  fakesize*read_read*haplotype_haplotype/computation_time/1000000000);
		return 0;
	}


